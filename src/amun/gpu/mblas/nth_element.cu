#include <iostream>
#include "common/utils.h"
#include "matrix_wrapper.h"
#include "nth_element.h"
#include "matrix_functions.h"

using namespace std;

namespace amunmt {
namespace GPU {

NthElement::NthElement(uint maxBeamSize, uint maxBatchSize)
: d_breakdown(maxBeamSize, 1, 1, 1)
, maxBeamSize_(maxBeamSize)
, maxBatchSize_(maxBatchSize)
{
  //cerr << "maxBatchSize=" << maxBatchSize << " maxBeamSize=" << maxBeamSize << endl;

  d_batchPosition.reserve(maxBatchSize + 1);
  d_cumBeamSizes.reserve(maxBatchSize + 1);

  d_res.reserve(maxBatchSize * maxBeamSize);
  h_res.reserve(maxBatchSize * maxBeamSize);
}

NthElement::~NthElement()
{
  //cerr << "FOO2" << endl;
}

void NthElement::getNBestList(const std::vector<uint>& beamSizes, mblas::Matrix& Probs, mblas::TMatrix<NthOut> &top,
                  std::vector<float>& outCosts, std::vector<uint>& outKeys,
                  const bool isFirst) {

  cerr << "top=" << top.Debug(2) << endl;
  /*
  cerr << "beamSizes=" << beamSizes.size() << endl;
  cerr << Debug(beamSizes, 2) << endl;
  cerr << "Probs=" << Probs.Debug(0) << endl;
  cerr << "outCosts=" << outCosts.size() << endl;
  cerr << "outKeys=" << outKeys.size() << endl;
  cerr << "isFirst=" << isFirst << endl;
  cerr << endl;
  */

  HostVector<uint> cummulatedBeamSizes(beamSizes.size() + 1);
  HostVector<uint> batchFirstElementIdxs(beamSizes.size() + 1);
  cummulatedBeamSizes[0] = 0;
  batchFirstElementIdxs[0] = 0;

  const uint vocabSize = Probs.dim(1);
  for (uint i = 0; i < beamSizes.size(); ++i) {

    cummulatedBeamSizes[i + 1] = cummulatedBeamSizes[i] + beamSizes[i];
    batchFirstElementIdxs[i + 1] = ((isFirst) ? (i + 1) : cummulatedBeamSizes[i + 1]) * vocabSize;
  }

  uint numHypos = cummulatedBeamSizes.back();
  d_res.NewSize(numHypos, 1, 1, 1);
  h_res.resize(numHypos);

  //cerr << endl;
  //cerr << "numHypos=" << numHypos << endl;
  //cerr << "beamSizes=" << Debug(beamSizes, 2) << endl;
  //cerr << "cummulatedBeamSizes=" << Debug(cummulatedBeamSizes, 2) << endl;
  //cerr << "batchFirstElementIdxs=" << Debug(batchFirstElementIdxs, 2) << endl;
  //cerr << "1Probs=" << Probs.Debug() << endl;

  getNBestList(Probs, batchFirstElementIdxs, cummulatedBeamSizes);

  //cerr << "2Probs=" << Probs.Debug() << endl;
  //cerr << "cummulatedBeamSizes.back()=" << cummulatedBeamSizes.back() << endl;
  //cerr << "cummulatedBeamSizes=" << Debug(cummulatedBeamSizes, 2) << endl;
  GetPairs(numHypos, outKeys, outCosts);

  //cerr << "outCosts=" << Debug(outCosts, 2) << endl;
  cerr << "outKeys=" << Debug(outKeys, 2) << endl;
  cerr << endl;
}

void NthElement::getNBestList(mblas::Matrix &probs,
                              const HostVector<uint>& batchFirstElementIdxs,
                              const HostVector<uint>& cummulatedBeamSizes)
{
  BEGIN_TIMER("getNBestList ex kernels");
  //std::cerr << "4Probs=" << probs.Debug(1) << std::endl;

  const uint vocabSize = probs.dim(1);
  const uint numBlocks = uint(maxBeamSize_ * vocabSize / (2 * BLOCK_SIZE)) + uint(maxBeamSize_ * vocabSize % (2 * BLOCK_SIZE) != 0);
  const uint numBatches = batchFirstElementIdxs.size() - 1;

  d_out.NewSize(maxBatchSize_ * numBlocks, 1, 1, 1);

  //cerr << "cummulatedBeamSizes=" << cummulatedBeamSizes.size() << endl;
  d_batchPosition.NewSize(batchFirstElementIdxs.size(), 1, 1, 1);
  d_cumBeamSizes.NewSize(cummulatedBeamSizes.size(), 1, 1, 1);
  assert(d_batchPosition.size() == d_cumBeamSizes.size());

  mblas::copy(thrust::raw_pointer_cast(batchFirstElementIdxs.data()),
              batchFirstElementIdxs.size(),
              d_batchPosition.data(),
              hipMemcpyHostToDevice);
  mblas::copy(thrust::raw_pointer_cast(cummulatedBeamSizes.data()),
              cummulatedBeamSizes.size(),
              d_cumBeamSizes.data(),
              hipMemcpyHostToDevice);

  mblas::MatrixWrapper<NthOut> outWrap(d_out);
  mblas::MatrixWrapper<float> probsWrap(probs);
  mblas::MatrixWrapper<uint> batchPositionWrap(d_batchPosition);
  mblas::MatrixWrapper<NthOut> resWrap(d_res, false);
  mblas::MatrixWrapper<uint> cumBeamSizesWrap(d_cumBeamSizes);

  PAUSE_TIMER("getNBestList ex kernels");

  BEGIN_TIMER("gMaxElement");
  gMaxElement<<<numBlocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(float), mblas::CudaStreamHandler::GetStream()>>>
    (outWrap, probsWrap, batchPositionWrap, numBatches);
  PAUSE_TIMER("gMaxElement");

  BEGIN_TIMER("gMaxElementUpdate");
  gMaxElementUpdate<<<numBatches, BLOCK_SIZE, BLOCK_SIZE * sizeof(float), mblas::CudaStreamHandler::GetStream()>>>
    (outWrap,
     probsWrap,
     resWrap,
     batchPositionWrap,
     cumBeamSizesWrap,
     numBlocks);
  PAUSE_TIMER("gMaxElementUpdate");

  /*
  cerr << "numBlocks=" << numBlocks << endl;
  cerr << "numBatches=" << numBatches << endl;
  cerr << "threads=" << BLOCK_SIZE << endl;

  cerr << "outWrap=" << outWrap.Debug() << endl;

  cerr << "probsWrap=" << probsWrap.Debug() << endl;

  cerr << "batchPositionWrap=" << batchPositionWrap.Debug() << endl;
  cerr << mblas::Debug(d_batchPosition, 2) << endl;

  cerr << "resWrap=" << resWrap.Debug() << endl;
  cerr << mblas::Debug(d_res, 2) << endl;

  cerr << "cumBeamSizesWrap=" << cumBeamSizesWrap.Debug() << endl;
  //cerr << mblas::Debug(d_cumBeamSizes, 2) << endl;

  cerr << endl;
  */
}

void NthElement::GetPairs(uint number,
                    std::vector<uint>& outKeys,
                    std::vector<float>& outValues)
{
  mblas::copy(d_res.data(), d_res.size(), thrust::raw_pointer_cast(h_res.data()), hipMemcpyDeviceToHost);
  BEGIN_TIMER("hipStreamSynchronize");
  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()) );
  PAUSE_TIMER("hipStreamSynchronize");

  for (uint i = 0; i < number; ++i) {
    outKeys.push_back(h_res[i].ind);
    outValues.push_back(h_res[i].score);
  }
}

void NthElement::getValueByKey(std::vector<float>& out, const mblas::Matrix &d_in) const
{
  // need a model with multiple scorers to test this method
  assert(false);

  mblas::MatrixWrapper<float> breakdownWrap(d_breakdown);
  const mblas::MatrixWrapper<float> inWrap(d_in);

  //gGetValueByKey<<<1, lastN_, 0, stream_>>>
  //  (breakdownWrap, inWrap, h_res_idx, lastN_);

  HANDLE_ERROR( hipMemcpyAsync(out.data(), d_breakdown.data(), h_res.size() * sizeof(float),
                                hipMemcpyDeviceToHost, mblas::CudaStreamHandler::GetStream()) );

  BEGIN_TIMER("cudaStreamSynchronize2");
  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
  PAUSE_TIMER("cudaStreamSynchronize2");
}

}  // namespace GPU
} // namespace amunmt
